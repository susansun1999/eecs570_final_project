// cd /home/hork/cuda-workspace/CudaSHA256/Debug/files
// time ~/Dropbox/FIIT/APS/Projekt/CpuSHA256/a.out -f ../file-list
// time ../CudaSHA256 -f ../file-list


#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
//#include "sha256.cuh"
#include <dirent.h>
#include <ctype.h>
#include <time.h>
#include <sys/time.h>
//#include <wb.h>

#define BLOCKSIZE 1024
#define BLOCKNUM  64
#define ROTLEFT(a,b) (((a) << (b)) | ((a) >> (32-(b))))
#define ROTRIGHT(a,b) (((a) >> (b)) | ((a) << (32-(b))))
#define EP0(x) (ROTRIGHT(x,2) ^ ROTRIGHT(x,13) ^ ROTRIGHT(x,22))
#define EP1(x) (ROTRIGHT(x,6) ^ ROTRIGHT(x,11) ^ ROTRIGHT(x,25))
#define CH(x,y,z) (((x) & (y)) ^ (~(x) & (z)))
#define MAJ(x,y,z) (((x) & (y)) ^ ((x) & (z)) ^ ((y) & (z)))


const uint32_t constH[8] = {0x6a09e667,0xbb67ae85,0x3c6ef372,0xa54ff53a,0x510e527f,0x9b05688c,0x1f83d9ab,0x5be0cd19};

const uint32_t constK[64] = {0x428a2f98, 0x71374491, 0xb5c0fbcf, 0xe9b5dba5, 0x3956c25b, 0x59f111f1, 0x923f82a4, 0xab1c5ed5,
   0xd807aa98, 0x12835b01, 0x243185be, 0x550c7dc3, 0x72be5d74, 0x80deb1fe, 0x9bdc06a7, 0xc19bf174,
   0xe49b69c1, 0xefbe4786, 0x0fc19dc6, 0x240ca1cc, 0x2de92c6f, 0x4a7484aa, 0x5cb0a9dc, 0x76f988da,
   0x983e5152, 0xa831c66d, 0xb00327c8, 0xbf597fc7, 0xc6e00bf3, 0xd5a79147, 0x06ca6351, 0x14292967,
   0x27b70a85, 0x2e1b2138, 0x4d2c6dfc, 0x53380d13, 0x650a7354, 0x766a0abb, 0x81c2c92e, 0x92722c85,
   0xa2bfe8a1, 0xa81a664b, 0xc24b8b70, 0xc76c51a3, 0xd192e819, 0xd6990624, 0xf40e3585, 0x106aa070,
   0x19a4c116, 0x1e376c08, 0x2748774c, 0x34b0bcb5, 0x391c0cb3, 0x4ed8aa4a, 0x5b9cca4f, 0x682e6ff3,
   0x748f82ee, 0x78a5636f, 0x84c87814, 0x8cc70208, 0x90befffa, 0xa4506ceb, 0xbef9a3f7, 0xc67178f2};

const uint32_t constIn[64*BLOCKSIZE*BLOCKNUM] = {0};

__global__ void pre_sha256_cuda(uint32_t* W){
  unsigned tx = blockIdx.x * BLOCKSIZE + threadIdx.x;
  int startingIdx = 64*tx + 16;
  int endingIdx = 64*tx + 64;
  for(unsigned i = startingIdx; i < endingIdx; i++){
    uint32_t s0 = ROTRIGHT(W[i-15],7) xor ROTRIGHT(W[i-15],18) xor ROTRIGHT(W[i-15],3);
    uint32_t s1 = ROTRIGHT(W[i-2],17) xor ROTRIGHT(W[i-2],19) xor ROTRIGHT(W[i-2],10);
    W[i] = W[i-16] + s0 + W[i-7] + s1;
  }
}

__global__ void sha256_cuda(uint32_t*K, uint32_t*H, uint32_t* W, unsigned times){
  unsigned tx = blockIdx.x * BLOCKSIZE + threadIdx.x;
  unsigned startingIdx = 64 * tx;

  for(unsigned it = 0; it < times; it++){
    uint32_t a = H[8*tx + 0];
    uint32_t b = H[8*tx + 1];
    uint32_t c = H[8*tx + 2];
    uint32_t d = H[8*tx + 3];
    uint32_t e = H[8*tx + 4];
    uint32_t f = H[8*tx + 5];
    uint32_t g = H[8*tx + 6];
    uint32_t h = H[8*tx + 7];
    
    for (unsigned i = 0; i < 64; ++i) {
      uint32_t t1 = h + EP1(e) + CH(e, f, g) + K[i] + W[startingIdx + i];
      uint32_t t2 = EP0(a) + MAJ(a, b, c);
      h = g;
      g = f;
      f = e;
      e = d + t1;
      d = c;
      c = b;
      b = a;
      a = t1 + t2;
    }
    H[8*tx + 0] += a;
    H[8*tx + 1] += b;
    H[8*tx + 2] += c;
    H[8*tx + 3] += d;
    H[8*tx + 4] += e;
    H[8*tx + 5] += f;
    H[8*tx + 6] += g;
    H[8*tx + 7] += h;
  }
}

int main(int argc, char **argv) {
  unsigned times = atoi(argv[1]);
  uint32_t* hostH = (uint32_t*)malloc(8*BLOCKSIZE*BLOCKNUM*sizeof(uint32_t));
  uint32_t* hostK = (uint32_t*)malloc(64*sizeof(uint32_t));
  uint32_t* hostIn = (uint32_t*)malloc(BLOCKSIZE*64*BLOCKNUM*sizeof(uint32_t));
  for(unsigned i = 0; i < BLOCKNUM*BLOCKSIZE*8; i++){
      hostH[i] = constH[i%8];
  }

  for(unsigned i = 0; i < 64; i++){
    hostK[i] = constK[i];
  }
  for(unsigned i = 0; i < BLOCKNUM*BLOCKSIZE*64; i++){
    hostIn[i] = constIn[i];
  }

  uint32_t* deviceH, *deviceK;
  uint32_t* deviceIn; // W

  hipMalloc(&deviceH, 8*BLOCKSIZE*BLOCKNUM*sizeof(uint32_t));
  hipMalloc(&deviceK, 64*sizeof(uint32_t));
  hipMalloc(&deviceIn, 64*BLOCKSIZE*BLOCKNUM*sizeof(uint32_t));

  hipMemcpy(deviceH,hostH,8*BLOCKSIZE*BLOCKNUM*sizeof(uint32_t),hipMemcpyHostToDevice);
  hipMemcpy(deviceK,hostK,64*sizeof(uint32_t),hipMemcpyHostToDevice);
  hipMemcpy(deviceIn,hostIn,64*BLOCKSIZE*BLOCKNUM*sizeof(uint32_t),hipMemcpyHostToDevice);

  unsigned numBlocks = BLOCKNUM;
  unsigned blockSize = BLOCKSIZE; // warp size

  pre_sha256_cuda <<< numBlocks, blockSize >>> (deviceIn);
  hipDeviceSynchronize();

  /* get start timestamp */
  struct timeval tv;
  gettimeofday(&tv,NULL);
  uint64_t start = tv.tv_sec*(uint64_t)1000000+tv.tv_usec;

  sha256_cuda <<< numBlocks, blockSize >>> (deviceK, deviceH, deviceIn,times);
  hipDeviceSynchronize();

 /* get elapsed time */
 gettimeofday(&tv,NULL);
 uint64_t end = tv.tv_sec*(uint64_t)1000000+tv.tv_usec;
 uint64_t elapsed = end - start;

 printf("it: %d @@@ Elapsed time (usec): %lld\n",times, elapsed);

 hipMemcpy(hostH,deviceH,8*BLOCKSIZE*BLOCKNUM*sizeof(uint32_t),hipMemcpyDeviceToHost);
/*
  for(unsigned i = 0; i < BLOCKSIZE*BLOCKNUM; i++){
    //hostIn[i] = constIn[i];
    for(unsigned j = 0; j < 8; j++){
      printf("%x ",hostH[i*8+j]);
    }
    printf("\n");
  }
  printf("\n");
*/

  hipFree(deviceH);
  hipFree(deviceK);
  hipFree(deviceIn);

  free(hostH);
  free(hostIn);
  free(hostK);
  return 0;
}