#include "hip/hip_runtime.h"
// cd /home/hork/cuda-workspace/CudaSHA256/Debug/files
// time ~/Dropbox/FIIT/APS/Projekt/CpuSHA256/a.out -f ../file-list
// time ../CudaSHA256 -f ../file-list


#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
//#include "sha256.cuh"
#include <dirent.h>
#include <ctype.h>
#include <time.h>
#include <sys/time.h>
#include <wb.h>

#define BLOCKSIZE 32
#define ROTLEFT(a,b) (((a) << (b)) | ((a) >> (32-(b))))
#define ROTRIGHT(a,b) (((a) >> (b)) | ((a) << (32-(b))))
#define EP0(x) (ROTRIGHT(x,2) ^ ROTRIGHT(x,13) ^ ROTRIGHT(x,22))
#define EP1(x) (ROTRIGHT(x,6) ^ ROTRIGHT(x,11) ^ ROTRIGHT(x,25))
#define CH(x,y,z) (((x) & (y)) ^ (~(x) & (z)))
#define MAJ(x,y,z) (((x) & (y)) ^ ((x) & (z)) ^ ((y) & (z)))


const uint32_t constH[8] = {0x6a09e667,0xbb67ae85,0x3c6ef372,0xa54ff53a,0x510e527f,0x9b05688c,0x1f83d9ab,0x5be0cd19};

const uint32_t constK[64] = {0x428a2f98, 0x71374491, 0xb5c0fbcf, 0xe9b5dba5, 0x3956c25b, 0x59f111f1, 0x923f82a4, 0xab1c5ed5,
   0xd807aa98, 0x12835b01, 0x243185be, 0x550c7dc3, 0x72be5d74, 0x80deb1fe, 0x9bdc06a7, 0xc19bf174,
   0xe49b69c1, 0xefbe4786, 0x0fc19dc6, 0x240ca1cc, 0x2de92c6f, 0x4a7484aa, 0x5cb0a9dc, 0x76f988da,
   0x983e5152, 0xa831c66d, 0xb00327c8, 0xbf597fc7, 0xc6e00bf3, 0xd5a79147, 0x06ca6351, 0x14292967,
   0x27b70a85, 0x2e1b2138, 0x4d2c6dfc, 0x53380d13, 0x650a7354, 0x766a0abb, 0x81c2c92e, 0x92722c85,
   0xa2bfe8a1, 0xa81a664b, 0xc24b8b70, 0xc76c51a3, 0xd192e819, 0xd6990624, 0xf40e3585, 0x106aa070,
   0x19a4c116, 0x1e376c08, 0x2748774c, 0x34b0bcb5, 0x391c0cb3, 0x4ed8aa4a, 0x5b9cca4f, 0x682e6ff3,
   0x748f82ee, 0x78a5636f, 0x84c87814, 0x8cc70208, 0x90befffa, 0xa4506ceb, 0xbef9a3f7, 0xc67178f2};

const uint32_t constIn[64*BLOCKSIZE] = {0};

__global__ void pre_sha256_cuda(uint32_t* W){
  int startingIdx = 64*threadIdx.x + 16;
  int endingIdx = 64*threadIdx.x + 64;
  for(unsigned i = startingIdx; i < endingIdx; i++){
    uint32_t s0 = ROTRIGHT(W[i-15],7) xor ROTRIGHT(W[i-15],18) xor ROTRIGHT(W[i-15],3);
    uint32_t s1 = ROTRIGHT(W[i-2],17) xor ROTRIGHT(W[i-2],19) xor ROTRIGHT(W[i-2],10);
    W[i] = W[i-16] + s0 + W[i-7] + s1;
  }
}

__global__ void sha256_cuda(uint32_t*K, uint32_t*H, uint32_t* W, uint32_t* out){
  uint32_t a = H[0];
  uint32_t b = H[1];
  uint32_t c = H[2];
  uint32_t d = H[3];
  uint32_t e = H[4];
  uint32_t f = H[5];
  uint32_t g = H[6];
  uint32_t h = H[7];
  unsigned startingIdx = 64 * threadIdx.x;
  for (unsigned i = 0; i < 64; ++i) {
		uint32_t t1 = h + EP1(e) + CH(e, f, g) + K[i] + W[startingIdx + i];
		uint32_t t2 = EP0(a) + MAJ(a, b, c);
		h = g;
		g = f;
		f = e;
		e = d + t1;
		d = c;
		c = b;
		b = a;
		a = t1 + t2;
	}
  //out[threadIdx.x*8] = H[0] + a;
  //out[threadIdx.x*8+1] = H[1] + b;
  //out[threadIdx.x*8+2] = H[2] + c;
  //out[threadIdx.x*8+3] = H[3] + d;
  //out[threadIdx.x*8+4] = H[4] + e;
  //out[threadIdx.x*8+5] = H[5] + f;
  //out[threadIdx.x*8+6] = H[6] + g;
  //out[threadIdx.x*8+7] = H[7] + h;
  out[threadIdx.x*8] = 110;
  out[threadIdx.x*8+1] = 110;
  out[threadIdx.x*8+2] = 110;
  out[threadIdx.x*8+3] = 110;
  out[threadIdx.x*8+4] = 110;
  out[threadIdx.x*8+5] = 110;
  out[threadIdx.x*8+6] = 110;
  out[threadIdx.x*8+7] = 110;
}

int main(int argc, char **argv) {
  wbArg_t args;

  uint32_t* hostH = (uint32_t*)malloc(8*sizeof(uint32_t));
  uint32_t* hostK = (uint32_t*)malloc(64*sizeof(uint32_t));
  uint32_t* hostIn = (uint32_t*)malloc(BLOCKSIZE*64*sizeof(uint32_t));
  uint32_t* hostOut = (uint32_t*)malloc(BLOCKSIZE*8*sizeof(uint32_t));
  for(unsigned i = 0; i < 8; i++){
    hostH[i] = constH[i];
  }
  for(unsigned i = 0; i < 64; i++){
    hostK[i] = constK[i];
  }
  for(unsigned i = 0; i < BLOCKSIZE*64; i++){
    //hostIn[i] = constIn[i];
    hostIn[i] = 0x78a5636f;
  }

  uint32_t* deviceH, *deviceK;
  uint32_t* deviceIn; // W
  uint32_t* deviceOut; // H

  hipMalloc(&deviceH, 8*sizeof(uint32_t));
  hipMalloc(&deviceK, 64*sizeof(uint32_t));
  hipMalloc(&deviceIn, 64*BLOCKSIZE*sizeof(uint32_t));
  hipMalloc(&deviceOut, 8*BLOCKSIZE*sizeof(uint32_t));

  hipMemcpy(deviceH,hostH,8*sizeof(uint32_t),hipMemcpyHostToDevice);
  hipMemcpy(deviceK,hostK,64*sizeof(uint32_t),hipMemcpyHostToDevice);
  hipMemcpy(deviceIn,hostIn,64*BLOCKSIZE*sizeof(uint32_t),hipMemcpyHostToDevice);

  unsigned numBlocks = 1;
  unsigned blockSize = BLOCKSIZE; // warp size

  pre_sha256_cuda <<< numBlocks, blockSize >>> (deviceIn);
  /* get start timestamp */
  struct timeval tv;
  gettimeofday(&tv,NULL);
  uint64_t start = tv.tv_sec*(uint64_t)1000000+tv.tv_usec;

  sha256_cuda <<< numBlocks, blockSize >>> (deviceK, deviceH, deviceIn, deviceOut);
  hipDeviceSynchronize();

  /* get elapsed time */
  gettimeofday(&tv,NULL);
  uint64_t end = tv.tv_sec*(uint64_t)1000000+tv.tv_usec;
  uint64_t elapsed = end - start;

  printf("@@@ Elapsed time (usec): %lld\n", elapsed);

  hipMemcpy(hostOut,deviceOut,8*BLOCKSIZE*sizeof(uint32_t),hipMemcpyDeviceToHost);

  for(unsigned i = 0; i < BLOCKSIZE; i++){
    //hostIn[i] = constIn[i];
    for(unsigned j = 0; j < 8; j++){
      printf("%x ",hostOut[i*BLOCKSIZE+j]);
    }
    printf("\n");
  }
  wbTime_start(GPU, "Freeing GPU Memory");

  hipFree(deviceOut);
  hipFree(deviceH);
  hipFree(deviceK);
  hipFree(deviceOut);

  wbTime_stop(GPU, "Freeing GPU Memory");

  wbSolution(args, hostOut, 8*BLOCKSIZE);

  return 0;
}
