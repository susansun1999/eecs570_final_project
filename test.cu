
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdio>
#include <cmath>
#include <math.h>
#include <float.h>
#define BLOCKSIZE 1024
#define BLOCKDIM 32
using namespace std;

__device__ double find2Smallest(double arr[], int arr_size)  
{  
    int i;
    double first = DBL_MAX, second = DBL_MAX;
  
    for (i = 0; i < arr_size ; i ++) {  
        // If current element is smaller than first then update both first and second
        if (arr[i] < first) {  
            second = first;  
            first = arr[i];  
        }  
        // If arr[i] is in between first and second then update second
        else if (arr[i] < second && arr[i] != first)  
            second = arr[i];  
    }  
    return second;
} 

__global__ void MatUpdate(double *dev_A, double *dev_new_A, const int n){
    __shared__ int neighbor[2];
    // double cmp[4];
    neighbor[0] = -1;
    neighbor[1] = 1;
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    int j = threadIdx.y + blockDim.y * blockIdx.y;
    
    if(i > 0 && i < n-1 && j > 0 && j < n-1) {
        double small_1 = DBL_MAX;
        double small_2 = DBL_MAX;
        for (int p : neighbor){
            for (int q : neighbor){
                double x = dev_A[(i + p) * n + (j + q)];
                if (x <= small_1){
                    small_2 = small_1;
                    small_1 = x;
                }
                else if (x < small_2){
                    small_2 = x;
                }
            }
        }
        dev_new_A[i * n + j] = dev_A[i * n + j] + small_2;
    }
}

__global__ void iterationKernel(double* A, double* new_A, size_t n)
{
    __shared__ double sdata[BLOCKSIZE + 4 * BLOCKDIM + 4];
    double cmp[4];
    int row = threadIdx.y + blockIdx.y * blockDim.y;    // global index
	int col = threadIdx.x + blockIdx.x * blockDim.x;
    int local_index = threadIdx.y * blockDim.x + threadIdx.x;
    size_t global_index = row * n + col;

    // load input into __shared__ memory
    sdata[local_index] = 0;
    if (global_index < n * n) {
        sdata[local_index] = A[global_index];

        // Except for blocks in the first col, threads in the first col load their left elements
        if (threadIdx.x == 0 && blockIdx.x != 0) {
            sdata[BLOCKSIZE + threadIdx.y] = A[global_index - 1];
            if (threadIdx.y == 0 && blockIdx.y != 0) sdata[BLOCKSIZE + 4 * BLOCKDIM] = A[global_index - n - 1]; // left up corner
        }
        // Except for blocks in the last col, threads in the last col load their right elements
        if (threadIdx.x == (blockDim.x - 1) && blockIdx.x != (gridDim.x - 1)) {
            sdata[BLOCKSIZE + BLOCKDIM + threadIdx.y] = A[global_index + 1];
            if (threadIdx.y == (blockDim.y - 1) && blockIdx.y != (gridDim.y - 1)) sdata[BLOCKSIZE + 4 * BLOCKDIM + 1] = A[global_index + n + 1]; // right down corner
        }
        // Except for blocks in the first row, threads in the first row load the elements above
        if (threadIdx.y == 0 && blockIdx.y != 0) {
            sdata[BLOCKSIZE + BLOCKDIM * 2 + threadIdx.x] = A[global_index - n];
            if (threadIdx.x == (blockDim.x - 1) && blockIdx.x != (gridDim.x - 1)) sdata[BLOCKSIZE + 4 * BLOCKDIM + 2] = A[global_index - n + 1]; // right up corner
        }
        // Except for blocks in the last row, threads in the last row load the elements below
        if (threadIdx.y == (blockDim.y - 1) && blockIdx.y != (gridDim.y - 1)) {
            sdata[BLOCKSIZE + BLOCKDIM * 3 + threadIdx.x] = A[global_index + n];
            if (threadIdx.x == 0 && blockIdx.x != 0) sdata[BLOCKSIZE + 4 * BLOCKDIM + 3] = A[global_index + n - 1]; // left down corner
        }
    }
    __syncthreads();        // wait for each thread to load the value to shared memory

    // update
    if (row > 0 && col > 0 && row < (n - 1) && col < (n - 1)) {
        if (threadIdx.x == 0) {                             // first col
            if (threadIdx.y == 0) {
                cmp[0] = sdata[BLOCKSIZE + 4 * BLOCKDIM];                       // left up
                cmp[1] = sdata[BLOCKSIZE + threadIdx.y + 1];                    // left down
                cmp[2] = sdata[BLOCKSIZE + BLOCKDIM * 2 + threadIdx.x + 1];     // right up
                cmp[3] = sdata[local_index + blockDim.x + 1];                   // right down
            } else if (threadIdx.y == (blockDim.y - 1)) {
                cmp[0] = sdata[BLOCKSIZE + threadIdx.y - 1];                    // left up
                cmp[1] = sdata[BLOCKSIZE + 4 * BLOCKDIM + 3];                   // left down
                cmp[2] = sdata[local_index - blockDim.x + 1];                   // right up
                cmp[3] = sdata[BLOCKSIZE + BLOCKDIM * 3 + threadIdx.x + 1];     // right down
            } else {
                cmp[0] = sdata[BLOCKSIZE + threadIdx.y - 1];                    // left up
                cmp[1] = sdata[BLOCKSIZE + threadIdx.y + 1];                    // left down
                cmp[2] = sdata[local_index - blockDim.x + 1];                   // right up
                cmp[3] = sdata[local_index + blockDim.x + 1];                   // right down
            }
        } else if (threadIdx.x == (blockDim.x - 1)) {       // last col
            if (threadIdx.y == 0) {
                cmp[0] = sdata[BLOCKSIZE + BLOCKDIM * 2 + threadIdx.x - 1];     // left up
                cmp[1] = sdata[local_index + blockDim.x - 1];                   // left down
                cmp[2] = sdata[BLOCKSIZE + 4 * BLOCKDIM + 2];                   // right up
                cmp[3] = sdata[BLOCKSIZE + BLOCKDIM + threadIdx.y + 1];         // right down
            } else if (threadIdx.y == (blockDim.y - 1)) {
                cmp[0] = sdata[local_index - blockDim.x - 1];                   // left up
                cmp[1] = sdata[BLOCKSIZE + BLOCKDIM * 3 + threadIdx.x - 1];     // left down
                cmp[2] = sdata[BLOCKSIZE + BLOCKDIM + threadIdx.y - 1];         // right up
                cmp[3] = sdata[BLOCKSIZE + 4 * BLOCKDIM + 1];                   // right down
            } else {
                cmp[0] = sdata[local_index - blockDim.x - 1];                   // left up
                cmp[1] = sdata[local_index + blockDim.x - 1];                   // left down
                cmp[2] = sdata[BLOCKSIZE + BLOCKDIM + threadIdx.y - 1];         // right up
                cmp[3] = sdata[BLOCKSIZE + BLOCKDIM + threadIdx.y + 1];         // right down
            }
        } else if (threadIdx.y == 0) {                      // first row
            if (threadIdx.x != 0 && threadIdx.x != (blockDim.x - 1)) {
                cmp[0] = sdata[BLOCKSIZE + BLOCKDIM * 2 + threadIdx.x - 1];     // left up
                cmp[1] = sdata[local_index + blockDim.x - 1];                   // left down
                cmp[2] = sdata[BLOCKSIZE + BLOCKDIM * 2 + threadIdx.x + 1];     // right up
                cmp[3] = sdata[local_index + blockDim.x + 1];                   // right down
            }
        } else if (threadIdx.y == (blockDim.y - 1)) {       // last row
            if (threadIdx.x != 0 && threadIdx.x != (blockDim.x - 1)) {
                cmp[0] = sdata[local_index - blockDim.x - 1];                   // left up
                cmp[1] = sdata[BLOCKSIZE + BLOCKDIM * 3 + threadIdx.x - 1];     // left down
                cmp[2] = sdata[local_index - blockDim.x + 1];                   // right up
                cmp[3] = sdata[BLOCKSIZE + BLOCKDIM * 3 + threadIdx.x + 1];     // right down
            }
        } else {
            cmp[0] = sdata[local_index - blockDim.x - 1];                       // left up
            cmp[1] = sdata[local_index + blockDim.x - 1];                       // left down
            cmp[2] = sdata[local_index - blockDim.x + 1];                       // right up
            cmp[3] = sdata[local_index + blockDim.x + 1];                       // right down
        }
        new_A[global_index] = sdata[local_index] + find2Smallest(cmp, 4);
    }
}


__global__ void sumKernel(double* A, double* per_block_result, size_t n)
{
    __shared__ double sdata[BLOCKSIZE];
    size_t bid = blockIdx.y * gridDim.x + blockIdx.x;
    size_t local_index = threadIdx.y * blockDim.x + threadIdx.x;
    size_t global_index = bid * BLOCKSIZE + local_index;

    // load input into __shared__ memory
    sdata[local_index] = 0;
    if (global_index < n) sdata[local_index] = A[global_index];
    __syncthreads();        // wait for each thread to load the value to shared memory

    for( int stride = BLOCKSIZE / 2; stride > 0; stride >>= 1) {
        if (local_index < stride) sdata[local_index] += sdata[local_index + stride];
        __syncthreads();    // sum is stored in A[0]
    }
    if (local_index == 0) per_block_result[bid] = sdata[0];
}

__global__ void verificationKernel(double* A, double* A_37_47, size_t n)
{
    *A_37_47 = A[37 * n + 47];
}


int main(int argc, char** argv) {
    int n = atoi(argv[1]), t = atoi(argv[2]);
    // Initialize
    int size = n * n ;
    double* A = new double [size];
    double* col_val = new double [n];
    for (int j = 0; j < n; ++j){
      col_val[j] = sin(j);
    }
    for (int i = 0; i < n; ++i){
        double row_val = cos(2*i);
        for (int j = 0; j < n; ++j){
            A[i*n+j] = (1 + row_val + col_val[j])*(1 + row_val + col_val[j]);
        }
    }
    delete [] col_val;
    // for (int i = 0; i <n*n; ++i){
    //     cout<< A[i]<<endl;
    // }
    double A_37_47 = 0;
    double A_sum = 0;

    dim3 blockSize(BLOCKDIM, BLOCKDIM);
    dim3 gridSize((n + blockSize.x - 1) / blockSize.x, (n + blockSize.y - 1) / blockSize.y);

    // Copy from CPU to GPU
    double* dev_A = 0;
    double* dev_new_A = 0;
    double* dev_A_37_47 = 0;
    double* dev_A_sum = 0;
    double *dev_per_block_result;
    int num_block = gridSize.x * gridSize.y;
    int tmp_num_block = num_block > BLOCKSIZE ? num_block / BLOCKSIZE + 1: 1;  // assign block num to store partial sum if num_block is large than 1024
    hipMalloc((void**)&dev_per_block_result, (num_block + tmp_num_block + 1) * sizeof(double));
    hipMalloc((void**)&dev_A, size * sizeof(double));
    hipMalloc((void**)&dev_new_A, size * sizeof(double));
    hipMalloc((void**)&dev_A_37_47, sizeof(double));
    hipMalloc((void**)&dev_A_sum, sizeof(double));

    hipMemcpy(dev_A, A, size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_new_A, A, size * sizeof(double), hipMemcpyHostToDevice);

    // Set up timing
    hipEvent_t start, stop;
    float gpu_time = 0.0f;
    double *swap;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    for (int iter = 0; iter < t; iter++){
        MatUpdate<<<gridSize, blockSize>>>(dev_A, dev_new_A, n);
        hipDeviceSynchronize();
        swap = dev_A;
        dev_A = dev_new_A;
        dev_new_A = swap;
    }

    verificationKernel <<< 1, 1 >>> (dev_A, dev_A_37_47, n);
    sumKernel <<< gridSize, blockSize >>> (dev_A, dev_per_block_result, size);    
    hipDeviceSynchronize();
    sumKernel <<< tmp_num_block, blockSize >>> (dev_per_block_result, dev_per_block_result + num_block, num_block);
    if (num_block > BLOCKSIZE) sumKernel <<< 1, blockSize >>> (dev_per_block_result + num_block, dev_per_block_result + num_block + tmp_num_block, tmp_num_block);
    hipDeviceSynchronize();

    // Close timing
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpu_time, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipMemcpy((void*)&A_sum, (void*)(dev_per_block_result + num_block), sizeof(double), hipMemcpyDeviceToHost);
    if (num_block > BLOCKSIZE) hipMemcpy((void*)&A_sum, (void*)(dev_per_block_result + num_block + tmp_num_block), sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy((void*)A, (void*)dev_A, size * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(&A_37_47, dev_A_37_47, sizeof(double), hipMemcpyDeviceToHost);
    hipFree(dev_per_block_result);
    hipFree(dev_A);
    hipFree(dev_new_A);
    hipFree(dev_A_37_47);
    hipFree(dev_A_sum);
    
    cout << fixed;
    cout << "Sum: "<< A_sum << endl;
    cout << "A(37, 47): "<< A_37_47 << endl;
    cout<<"Time: "<<gpu_time<<endl;
    return 0;
}
